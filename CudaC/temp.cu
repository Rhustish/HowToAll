

#include <hip/hip_runtime.h>
#include<stdio.h>
void helloCPU(){
    printf("Hello from CPU\n");
}

__global__ void helloGPU(){
    printf("Hello from GPU\n");
}

__global__ void add(int a , int b , int *c){
    *c = a + b;
}

int main(){
    // helloCPU();
    // helloGPU<<<1,8>>>();
    // cudaDeviceSynchronize();

    // float *d_A;
    // int n; scanf("%d",&n);
    // int size = n*sizeof(float)*100;
    // cudaMalloc((void**)&d_A,size);
    // cudaFree(d_A);
    
    int a , b, c;
    int *dev_c;
    a=3;
    b=4;
    hipMalloc((void**)&dev_c,sizeof(int));
    //1024 is the max no. of GPU threads
    //if you want more, increase the no of block count(on the left)
    add<<<1,1024>>>(a,b,dev_c);
    // cudaDeviceSynchronize();
    hipMemcpy(&c , dev_c , sizeof(int),hipMemcpyDeviceToHost);
    printf("%d + %d = %d",a,b,c);
    hipFree(dev_c);
    

    return 0;
}